/**************************************************************************
 Name        : dnls.cu
 Author      : J.D. Bodyfelt
 Version     :
 Copyright   : (c) 2016, Massey University
 Description :
 **************************************************************************/

#include "dnls.cuh"

int main(void)
{
	hipSetDevice(1);
	Time tobj(1e-2, 1e3, false);
	Lattice DNLS(0.0f, 4.0f);
	std::string fbase("Test");
	DNLS.Evolve(tobj, fbase);

	return 0;
}

